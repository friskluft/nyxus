#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <builtin_types.h>
#include <iostream>

bool gpu_initialize(int dev_id)
{
	// Are there any GPU devices?
	int nDevices;
	hipGetDeviceCount(&nDevices);
	if (nDevices < 1)
		return false;

	// Establish the context
	if (hipSetDevice(dev_id) != hipSuccess)
		return false;

	return true;
}
